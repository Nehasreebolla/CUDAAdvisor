#include "hip/hip_runtime.h"
// add_vector.cu

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "my.cu"


#define CHECK_CUDA_ERROR(call)                                           \
    do {                                                                 \
        hipError_t err = call;                                          \
        if (err != hipSuccess) {                                        \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << " - " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE);                                          \
        }                                                                \
    } while (0)

extern "C" __global__ void addVectors(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n){
        c[i] = a[i] + b[i];
    
        if (c[i] > 10){
            a[i] = 10;
        }
        else {
            a[i] = 11;
        }

        for (int j = 0; j < 10; ++j) {
            a[i] += 1;
            c[i] = c[i]*c[i];
        }

        b[i] = 2;
    }
    
}

int main() {
    int n = 10; // 1M elements
    size_t size = n * sizeof(float);

    // Allocate host memory
    float *h_a = new float[n];
    float *h_b = new float[n];
    float *h_c = new float[n];

    // Initialize input vectors
    for (int i = 0; i < n; ++i) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    CHECK_CUDA_ERROR(hipMalloc(&d_a, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_b, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_c, size));

    // Copy data from host to device
    CHECK_CUDA_ERROR(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Check for kernel launch errors and sync errors
    CHECK_CUDA_ERROR(hipGetLastError());   // Kernel launch
    CHECK_CUDA_ERROR(hipDeviceSynchronize()); // Wait for kernel to finish

    // Copy result from device to host
    CHECK_CUDA_ERROR(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));

    // Verify result
    for (int i = 0; i < 10; ++i)
        std::cout << "h_c[" << i << "] = " << h_c[i] << std::endl;

    // Cleanup
    CHECK_CUDA_ERROR(hipFree(d_a));
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_c));
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
