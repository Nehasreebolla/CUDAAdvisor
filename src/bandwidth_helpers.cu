
#include <hip/hip_runtime.h>
// Device variables
__device__ unsigned long long total_bytes_accessed = 0;
__device__ unsigned long long total_memory_time_ns = 0;

// Bandwidth measurement functions
extern "C" {
    __device__ unsigned long long getGpuTime() {
        return clock64();
    }

    __device__ __noinline__ void recordMemAccess(unsigned long long bytes, unsigned long long cycles) {
        atomicAdd(&total_bytes_accessed, bytes);
        atomicAdd(&total_memory_time_ns, cycles);
    }

}

extern "C" void computeBandwidth() {
    unsigned long long bytes, time_ns;
    hipMemcpyFromSymbol(&bytes, HIP_SYMBOL(total_bytes_accessed), sizeof(unsigned long long));
    hipMemcpyFromSymbol(&time_ns, HIP_SYMBOL(total_memory_time_ns), sizeof(unsigned long long));
    double bandwidth = (bytes/(time_ns * 1e-9));
	printf("Memory Bandwidth: %.2f GB/s\n", bandwidth);
    printf("Total Bytes Accessed: %llu\n", bytes);
    printf("Total Time: %.3f ms\n", time_ns * 1000.0);
}


