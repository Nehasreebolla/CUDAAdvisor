
#include <hip/hip_runtime.h>
// Device variables
__device__ unsigned long long total_bytes_accessed = 0;
__device__ unsigned long long total_memory_time_ns = 0;

// Bandwidth measurement functions
extern "C" {
    __device__ unsigned long long getGpuTime() {
        return clock64();
    }

    __device__ __noinline__ void recordMemAccess(unsigned long long bytes, unsigned long long cycles) {
        atomicAdd(&total_bytes_accessed, bytes);
        atomicAdd(&total_memory_time_ns, cycles);
    }

}

extern "C" void computeBandwidth() {
    unsigned long long bytes, time_ns;
    hipMemcpyFromSymbol(&bytes, HIP_SYMBOL(total_bytes_accessed), sizeof(unsigned long long));
    hipMemcpyFromSymbol(&time_ns, HIP_SYMBOL(total_memory_time_ns), sizeof(unsigned long long));
    double bandwidth = ((double)bytes / 1e9) / (time_ns * 1e-9);  // GB/s
    
    printf("Memory Bandwidth: %.8f GB/cycles\n", bandwidth);
    printf("Total Bytes Accessed: %llu bytes\n", bytes);
    printf("Total Time: %.8llu cycles\n", time_ns);  // ns 
}


