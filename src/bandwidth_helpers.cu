#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "bandwidth_helpers.h"

// Define the device variables here (only once in the project)
__device__ unsigned long long total_bytes = 0;
__device__ unsigned long long total_time_ns = 0;

// Rest of your implementation...
extern "C" __device__ unsigned long long getGpuTime() {
    return clock64();
}

extern "C" __device__ void recordMemAccess(unsigned long long bytes, unsigned long long time) {
    atomicAdd(&total_bytes, bytes);
    atomicAdd(&total_time_ns, time);
}

__host__ void computeBandwidth() {
    unsigned long long bytes, time_ns;
    hipMemcpyFromSymbol(&bytes, HIP_SYMBOL(total_bytes), sizeof(unsigned long long));
    hipMemcpyFromSymbol(&time_ns, HIP_SYMBOL(total_time_ns), sizeof(unsigned long long));
    if (time_ns > 0) {
        double bandwidth = (double)bytes / (time_ns * 1e-9) / 1e9; // GB/s
        printf("Bandwidth: %.2f GB/s\n", bandwidth);
    } else {
        printf("No bandwidth data available\n");
    }
}