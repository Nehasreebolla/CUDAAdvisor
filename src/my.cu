
#include <hip/hip_runtime.h>
#include <stdio.h>

#ifdef __CUDA_ARCH__
// Hello world device function
__attribute__((noinline))
extern "C" __device__ void print(){

    printf("Hello world from device function!!\n");
}
#else

__attribute__((noinline))
extern "C" __host__ void print(){

    printf("Hello World from host function!!\n");
}
#endif