#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "bandwidth_helpers.cu"
#include "compute_intensity.cu"

// Device helper functions
__device__ float foo_CC(float a) {
    return a * 0.9;
}

__device__ int foo_DD(float a) {
    if (threadIdx.x < 2 || threadIdx.y > 2)
        return (int)a;
    else
        return a + 2;
}

__device__ float foo_BB(float a) {
    if (threadIdx.x > 3 || threadIdx.y > 11)
        return a + foo_CC(a);
    else
        return a + (float)foo_DD(a) / 2;
}

__device__ float foo_AA(float a, float b) {
    if (threadIdx.x < 8 || threadIdx.y > 4)
        return a * 3.1415 + 1;
    else
        return (b * a) * 0.5 + foo_BB(b);
}

// Kernels (unchanged)
__global__ void axpy_kernel1(float a, float* x, float* y) {

	//getGpuTime();
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int index = threadId;
	bool i = 1;
	bool arr[10];
	arr[0] = i;
	bool b = arr[0];
	int shifted = b << 1; // shifted = 2 if b was true, 0 if false
	shifted = shifted + 1;
	char name[4];
	name[1] = 'a';
	name[2] = 'b';
	char c = name[2];
	c = c+1;

    y[index] = x[index] * 0.3;
    if (index > 2)
        y[index] += 99;
    else
        y[index] += 999 + foo_CC(a);
}

__global__ void axpy_kernel2(float a, float* x, float* y) {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int index = threadId;

    float aa = y[index] + x[index] + 1.1;
    float b = 0.5 * y[index] + 0.25 * x[index] + 1.0;
    y[index] += (x[index] * 1.67 + foo_AA(aa, b));
}

// Main function
int main(int argc, char* argv[]) {
    float a = 2.0f;
    hipSetDevice(0);

    if (argc != 5) {
        printf("usage: ./axpy [blocks_x] [blocks_y] [space]threads_x] [threads_y]\n");
        exit(1);
    }

    int blocksx = atoi(argv[1]);
    int blocksy = atoi(argv[2]);
    int kDataLenx = atoi(argv[3]);
    int kDataLeny = atoi(argv[4]);

    int sizen = blocksx * blocksy * kDataLenx * kDataLeny;
    hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 500);

    float* host_x = (float*)malloc(sizen * sizeof(float));
    float* host_y = (float*)malloc(sizen * sizeof(float));
    void* host_newbu = (void*)malloc(1000);

    for (int ii = 0; ii < sizen; ii++) {
        host_x[ii] = ii % 8;
        host_y[ii] = ii % 5;
    }

    float* device_x;
    float* device_y;
    hipMalloc((void**)&device_x, sizen * sizeof(float));
    hipMalloc((void**)&device_y, sizen * sizeof(float) + 18);

    hipMemcpy(device_x, host_x, sizen * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_y, host_y, sizen * sizeof(float), hipMemcpyHostToDevice);

    dim3 CTAs(blocksx, blocksy);
    dim3 Threads(kDataLenx, kDataLeny);
    std::cout << "launching kernel...\n";

    // Reset counters before kernel launch
    unsigned long long zero = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(total_bytes_accessed), &zero, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(total_memory_time_ns), &zero, sizeof(unsigned long long));

    axpy_kernel1<<<CTAs, Threads>>>(a, device_x, device_y);
    hipDeviceSynchronize();
	computeBandwidth();
	printComputeIntensity();

    axpy_kernel2<<<CTAs, Threads>>>(a, device_x, device_y);
    hipDeviceSynchronize();
    //computeBandwidth();

    hipMemcpy(host_y, device_y, sizen * sizeof(float), hipMemcpyDeviceToHost);

    free(host_newbu);
    hipFree(device_x);
    hipFree(device_y);

    int verify = 0;
    for (int ii = 0; ii < 8; ii++)
        std::cout << "y[" << ii << "] = " << host_y[ii] << "\n";

    for (int ii = 0; ii < sizen; ii++) {
        if (host_y[ii] == ii % 5)
            verify++;
    }
    std::cout << "\n\n[TOOL verify] There are a total of\t" << verify << " incorrect numbers." << std::endl;
    if (verify == 0)
        std::cout << "[TOOL verify] passed!" << std::endl << std::endl;

    hipDeviceReset();
    free(host_x);
    free(host_y);
    return 0;
}
