
#include <hip/hip_runtime.h>
#include <stdio.h>

// Device counters
__device__ unsigned long long total_flops = 0;
__device__ unsigned long long total_bytes = 0;

extern "C" __device__ void recordFlop(unsigned long long count) {
    atomicAdd(&total_flops, count);
}

extern "C" __device__ void recordBytesAccess(unsigned long long bytes) {
    atomicAdd(&total_bytes, bytes);
}

extern "C" void printComputeIntensity() {
    unsigned long long flops, bytes;
    hipMemcpyFromSymbol(&flops, HIP_SYMBOL(total_flops), sizeof(unsigned long long));
    hipMemcpyFromSymbol(&bytes, HIP_SYMBOL(total_bytes), sizeof(unsigned long long));

    printf("\nCompute Intensity Results:\n");
    printf("Total FLOPs: %llu\n", flops);
    printf("Total Bytes: %llu\n", bytes);
    if (bytes > 0) {
        printf("Compute Intensity: %.2f FLOPs/byte\n", (double)flops/bytes);
    }
    
    // Reset counters before kernel launch
    unsigned long long zero = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(total_bytes), &zero, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(total_flops), &zero, sizeof(unsigned long long));
}
